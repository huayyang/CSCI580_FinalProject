#include "hip/hip_runtime.h"
#include "rayTracingProcessor.cuh"
#include "defines.h"
#include "math_functions.h"
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <iostream>

__device__ unsigned int x = 123456789,
y = 362436000,
z = 521288629,
c = 7654321; /* Seed variables */

__device__ unsigned int KISS()
{
	unsigned long long t, A = 698769069ULL;

	x = 69069 * x + 12345;

	y ^= (y << 13); y ^= (y >> 17); y ^= (y << 5);

	t = (A*z + c);
	c = (t >> 32);
	z = t;
	//return cuRAND();
	return x + y + z;
}

__device__ float3 crossProduct(float3 a, float3 b)
{
	float3 result;
	result.x = a.y * b.z - a.z * b.y;
	result.y = a.z * b.x - a.x * b.z;
	result.z = a.x * b.y - a.y * b.x;

	return result;
}

__device__ float dotProduct(float3 a, float3 b)
{
	return a.x * b.x + a.y * b.y + a.z * b.z;
}

__device__ float3 normalize(float3 vector)
{
	float3 result;
	float value = (vector.x * vector.x + vector.y * vector.y + vector.z * vector.z);
	value = sqrtf(value);
	if (value < 0.001 && value > -0.001)
	{
		result.x = 0;
		result.y = 0;
		result.z = 0;
	}
	else
	{
		result.x = vector.x / value;
		result.y = vector.y / value;
		result.z = vector.z / value;
	}

	return result;
}

__device__ bool isInside(float3 point, float3* triangle)
{
	float3 pointTo0, pointTo1, pointTo2;
	float3 edge0to1, edge1to2, edge2to0;
	float3 cross0, cross1, cross2;
	float value0, value1, value2;

	pointTo0.x = triangle[0].x - point.x;
	pointTo0.y = triangle[0].y - point.y;
	pointTo0.z = triangle[0].z - point.z;

	pointTo1.x = triangle[1].x - point.x;
	pointTo1.y = triangle[1].y - point.y;
	pointTo1.z = triangle[1].z - point.z;

	pointTo2.x = triangle[2].x - point.x;
	pointTo2.y = triangle[2].y - point.y;
	pointTo2.z = triangle[2].z - point.z;

	edge0to1.x = triangle[1].x - triangle[0].x;
	edge0to1.y = triangle[1].y - triangle[0].y;
	edge0to1.z = triangle[1].z - triangle[0].z;

	edge1to2.x = triangle[2].x - triangle[1].x;
	edge1to2.y = triangle[2].y - triangle[1].y;
	edge1to2.z = triangle[2].z - triangle[1].z;

	edge2to0.x = triangle[0].x - triangle[2].x;
	edge2to0.y = triangle[0].y - triangle[2].y;
	edge2to0.z = triangle[0].z - triangle[2].z;

	cross0 = normalize(crossProduct(pointTo0, edge0to1));
	cross1 = normalize(crossProduct(pointTo1, edge1to2));
	cross2 = normalize(crossProduct(pointTo2, edge2to0));
	value0 = dotProduct(cross0, cross1);
	value1 = dotProduct(cross1, cross2);
	value2 = dotProduct(cross2, cross0);

	if ((value0 >= -0.001 && value1 >= -0.001 && value2 >= -0.001))
		return true;
	else
		return false;
}

__device__ float checkDis(float3* vertex, float3 pos, float3 dir)
{
	//step1 calculate normal
	float3 edge1, edge2, normal;
	edge1.x = vertex[1].x - vertex[0].x;
	edge1.y = vertex[1].y - vertex[0].y;
	edge1.z = vertex[1].z - vertex[0].z;

	edge2.x = vertex[2].x - vertex[1].x;
	edge2.y = vertex[2].y - vertex[1].y;
	edge2.z = vertex[2].z - vertex[1].z;

	normal = normalize(crossProduct(edge1, edge2));

	//step2 calculate the projected vector
	float3 linkEdge, projectedVector;
	linkEdge.x = vertex[0].x - pos.x;
	linkEdge.y = vertex[0].y - pos.y;
	linkEdge.z = vertex[0].z - pos.z;

	float projectedValue = -dotProduct(linkEdge, normal);
	projectedVector.x = -projectedValue * normal.x;
	projectedVector.y = -projectedValue * normal.y;
	projectedVector.z = -projectedValue * normal.z;

	//step3 calculate the intersected point
	float3 intersected;
	float projectedValueOntoLine = dotProduct(projectedVector, dir);
	if (projectedValueOntoLine <= 0)
		return MAX_DIS;

	float distance = projectedValue * projectedValue / projectedValueOntoLine;
	intersected.x = pos.x + distance * dir.x;
	intersected.y = pos.y + distance * dir.y;
	intersected.z = pos.z + distance * dir.z;

	//step4 check if intersected
	if (isInside(intersected, vertex))
		return distance;
	else
		return MAX_DIS;
}


// 
__device__ float hitSurface(float3* vertex, float3 pos, float3 dir, float3* pho)
{
	//step1 calculate normal
	float3 edge1, edge2, normal;
	edge1.x = vertex[1].x - vertex[0].x;
	edge1.y = vertex[1].y - vertex[0].y;
	edge1.z = vertex[1].z - vertex[0].z;

	edge2.x = vertex[2].x - vertex[1].x;
	edge2.y = vertex[2].y - vertex[1].y;
	edge2.z = vertex[2].z - vertex[1].z;

	normal = normalize(crossProduct(edge1, edge2));

	//step2 calculate the projected vector
	float3 linkEdge, projectedVector;
	linkEdge.x = vertex[0].x - pos.x;
	linkEdge.y = vertex[0].y - pos.y;
	linkEdge.z = vertex[0].z - pos.z;

	float projectedValue = -dotProduct(linkEdge, normal);
	projectedVector.x = -projectedValue * normal.x;
	projectedVector.y = -projectedValue * normal.y;
	projectedVector.z = -projectedValue * normal.z;

	//step3 calculate the intersected point
	float3 intersected;
	float projectedValueOntoLine = dotProduct(projectedVector, dir);
	if (projectedValueOntoLine <= 0)
		return MAX_DIS;

	float distance = projectedValue * projectedValue / projectedValueOntoLine;
	intersected.x = pos.x + distance * dir.x;
	intersected.y = pos.y + distance * dir.y;
	intersected.z = pos.z + distance * dir.z;

	//step4 check if intersected
	if (isInside(intersected, vertex))
	{
		pho->x = intersected.x;
		pho->y = intersected.y;
		pho->z = intersected.z;
		return distance;
	}
	else
		return MAX_DIS;
}


__device__ void swapValue(float &a, float &b){
	float temp = a;
	a = b;
	b = temp;
}

__device__ void splitSort(float *A, int n, int low, int high)
{
	if (low >= high)
		return;
	int left = low;
	int right = high;
	bool moveRight = true;
	while (left != right){
		if (moveRight){
			if (A[left] > A[right])
			{
				swapValue(A[left], A[right]);
				moveRight = false;
			}
			else
			{
				right--;
			}
		}
		else{
			if (A[left] > A[right])
			{
				swapValue(A[left], A[right]);
				moveRight = true;
			}
			else
			{
				left++;
			}
		}
	}
	splitSort(A, n, low, left - 1);
	splitSort(A, n, left + 1, high);
}

__device__ uchar4 getColor(int currentIndex, uchar4 * pixels, int count, float3* vertex, float3* normal, uchar4* color, Material* materials, uchar1* materialIndex, float3 pos, float3 dir, Photon* photons)
{
	uchar4 resultColor;

	float minDis = MAX_DIS;
	int index = -1;
	float3 hitpoint;

	for (int k = 0; k<count; k++)
	{
		if (k == currentIndex)
			continue;

		float3 hitPos;
		float distance = hitSurface(vertex + k * 3, pos, dir, &hitPos);
		if (distance < minDis && distance > 0.001)
		{
			minDis = distance;
			index = k;
			hitpoint.x = hitPos.x; hitpoint.y = hitPos.y; hitpoint.z = hitPos.z;
		}
	}

	resultColor.x = 0;
	resultColor.y = 0;
	resultColor.z = 0;

	if (index != -1)
	{
		//printf("%d\n",(int)(materialIndex[index].x));
		Material hitMat = materials[(int)(materialIndex[index].x)];
		float Kd = hitMat.Kd;
		float Ks = hitMat.Ks;
		float Kni = 1 - Kd - Ks;
		if (Kd > 0.001)
		{
			int radius = 50;
			float distances[100] = { 0 };
			for (int k = 0; k<100; k++)
			{
				float3 temp;
				temp.x = hitpoint.x - photons[k].pos.x;
				temp.y = hitpoint.y - photons[k].pos.y;
				temp.z = hitpoint.z - photons[k].pos.z;
				float dis = dotProduct(temp, temp);
				distances[k] = dis;
			}
			// sort and get the middle distance
			//splitSort(distances,100,0,99);

			//if (currentIndex != -1)
			//	printf("%d \n",currentIndex);
			resultColor.x += Kd * (color[index * 3].x / distances[radius] * 3000 > 255 ? 255 : color[index * 3].x / distances[radius] * 3000);
			resultColor.y += Kd * (color[index * 3].y / distances[radius] * 3000 > 255 ? 255 : color[index * 3].y / distances[radius] * 3000);
			resultColor.z += Kd * (color[index * 3].z / distances[radius] * 3000 > 255 ? 255 : color[index * 3].z / distances[radius] * 3000);
		}

		// sort and get the middle distance
		//splitSort(distances,100,0,99);

		if (Ks > 0.001)
		{
			float NdotDir = -dotProduct(normal[index * 3], dir);
			float3 reflectDir;
			reflectDir.x = normal[index * 3].x * 2 * NdotDir + dir.x;
			reflectDir.y = normal[index * 3].y * 2 * NdotDir + dir.y;
			reflectDir.z = normal[index * 3].z * 2 * NdotDir + dir.z;
			//printf("%d %f %f %f \n",currentIndex,reflectDir.x,reflectDir.y,reflectDir.z);
			//printf("%d %f %f %f \n",currentIndex,normal[index * 3].x,normal[index * 3].y,normal[index * 3].z);
			//printf("%d %f %f %f \n\n",currentIndex,dir.x,dir.y,dir.z);

			uchar4 speculateColor = getColor(index, pixels, count, vertex, normal, color, materials, materialIndex, hitpoint, reflectDir, photons);
			//currentIndex = index;
			//pos = hitpoint;
			//dir = reflectDir;
			//goto START;
			resultColor.x += Ks * speculateColor.x;
			resultColor.y += Ks * speculateColor.y;
			resultColor.z += Ks * speculateColor.z;

		}
	}

	return resultColor;
}


__global__ void kernel(int indexX,int indexY,int unitX,int unitY,uchar4 * pixels,int count,float3* vertex,float3* normal,uchar4* color,Material* materials,uchar1* materialIndex,unsigned int width,unsigned int height,Camera* cam,Photon* photons)
{
	int i = blockIdx.x + indexX * unitX;
	int j = threadIdx.x + indexY * unitY;
	int offsetX = i - width / 2;
	int offsetY = height / 2 - j;
	float3 dir;
	dir.x = cam->lookat.x + (cam->tan_fov_2 * 2 * offsetY / height) * cam->up.x + (cam->tan_fov_2 * 2 * offsetX / height) * cam->right.x;
	dir.y = cam->lookat.y + (cam->tan_fov_2 * 2 * offsetY / height) * cam->up.y + (cam->tan_fov_2 * 2 * offsetX / height) * cam->right.y;
	dir.z = cam->lookat.z + (cam->tan_fov_2 * 2 * offsetY / height) * cam->up.z + (cam->tan_fov_2 * 2 * offsetX / height) * cam->right.z;

	dir = normalize(dir);

	int id = i + j * width;
	
	pixels[id] = getColor(-1,pixels,count,vertex,normal,color,materials,materialIndex,cam->pos,dir,photons);
	
}

__global__ void CastPhoton(uchar4 * pixels, int count, float3* vertex, Photon* photons, float3 lightPos)
{
	int i = blockIdx.x;
	int j = threadIdx.x;
	float3 dir;

	if (i >= 10 || j >= 10)
		return;


	dir.x = photons[i * 10 + j].pos.x;
	dir.y = photons[i * 10 + j].pos.y;
	dir.z = -10;
	photons[i * 10 + j].pos.x = photons[i * 10 + j].pos.y = photons[i * 10 + j].pos.z = -100;
	dir = normalize(dir);

	float minDis = MAX_DIS;
	int index = -1;
	for (int k = 0; k<count; k++)
	{
		float3 temp;
		float distance = hitSurface(vertex + k * 3, lightPos, dir, &temp);
		if (distance < minDis)
		{
			minDis = distance;
			index = k;
			photons[i * 10 + j].pos.x = temp.x;
			photons[i * 10 + j].pos.y = temp.y;
			photons[i * 10 + j].pos.z = temp.z;
		}
	}
	if (index != -1)
	{
		photons[i * 10 + j].power.x = 255;
		photons[i * 10 + j].power.y = 255;
		photons[i * 10 + j].power.z = 255;
	}
	else
	{
		photons[i * 10 + j].power.x = 0;
		photons[i * 10 + j].power.y = 0;
		photons[i * 10 + j].power.z = 0;
	}
}

// Helper function for using CUDA to add vectors in parallel.
void rayTracingCuda(uchar4 * pixels, int count, float3* vertex, float3* normal, uchar4* color, Photon* photons, Material* materials, uchar1* materialIndex)
{
	dim3 photonBlock(10);
	dim3 photonThread(10);
	// compute light photons
	CastPhoton<<<photonBlock,photonThread>>>(pixels,count,vertex,photons,LIGHT_POS);
	hipDeviceSynchronize();  
	
	//Photon* photonBuffer = (Photon*)malloc(100 * sizeof(Photon));
	//hipMemcpy(photonBuffer,photons,100 * sizeof(Photon),hipMemcpyDeviceToHost);
	//
	//for(int i =0;i< 100;i++)
	//{
	//	std::cout<<" "<<photonBuffer[i].pos.x<<" "<<photonBuffer[i].pos.y<<" "<<photonBuffer[i].pos.z<<"\t";
	//}

	Camera* cam = (Camera*)malloc(sizeof(Camera));
	cam->pos = CAM_POS;
	cam->lookat = CAM_LOOKAT;
	cam->up = CAM_LOOKUP;
	cam->right = CAM_LOOKRIGHT;
	cam->fov = CAM_FOV;
	cam->tan_fov_2 = tan(cam->fov * PI /2 / 180);
	hipMalloc((void**)&mainCamera_CUDA,sizeof(Camera));
	hipMemcpy(mainCamera_CUDA,cam,sizeof(Camera),hipMemcpyHostToDevice);

	int width = SCR_WIDTH;
	int indexX = 0;
	while (width != 0)
	{
		int x;
		int height = SCR_HEIGHT;
		int indexY = 0;

		if (width > UNIT_X)
			x = UNIT_X;
		else
			x = width;

		while (height != 0)
		{
			int y;
			if (height > UNIT_Y)
				y = UNIT_Y;
			else
				y = height;
			
			dim3 dimblock(x);
			dim3 dimthread(y);

			// Launch a kernel on the GPU with one thread for each element.
			
			kernel<<<dimblock,dimthread>>>(indexX,indexY,UNIT_X,UNIT_Y,pixels,count,vertex,normal,color,materials,materialIndex,SCR_WIDTH,SCR_HEIGHT,mainCamera_CUDA,photons);

			hipDeviceSynchronize();

			height -= y;
			indexY++;
		}
		width -= x;
		indexX++;
	}

}
